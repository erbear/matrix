#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Matrix.h"
#include <windows.h>
#include <vector>

using namespace std;

__global__
void multiplyCRS(int *ptr, int *ind, double *data, double *vec, double *res, int N) {
	int row = blockDim.x * blockIdx.x + threadIdx.x;

	if ( row < N ){
		double suma = 0;
		for (int jj = ptr[row]; jj < ptr[row + 1]; jj++){
			int j = ind[jj];
			suma += data[jj] * vec[j];
		}
		res[row] += suma;
	}
}




int main()
{
	Matrix m;
	m.readFromMtx("bcsstm13.mtx");
	m.generateNewVector();
	m.multiplyMatrix();
	m.modifiedSparseCompression();

	int block_size = 256;
	int N = m.getSize();
	int I = m.getIndSize();
	int P = m.getPtrSize();
	int D = m.getDataSize();
	int grid_number = ceil(N / (float)block_size);
	int thread_number = grid_number * block_size;

	double summary = 0.0;
	double *vectord;
	double *resultd;
	int *ptrd; 
	int *indd; 
	double *datad;
	vector <double> check;
	double *vector = m.getVector();
	double *result = new double[N];
	int *ptr = m.getPtr();
	int *ind = m.getInd();
	double *data = m.getData();

	fill(result, result + N, 0.);

	// rozmiar
	const int vecor_size = N*sizeof(double);
	const int result_size = N*sizeof(double);
	const int data_size = D*sizeof(double);
	const int ptr_size = P*sizeof(int);
	const int ind_size = I*sizeof(int);

	// zalokowanie
	
	hipMalloc((void**)&ptrd, ptr_size);
	hipMalloc((void**)&indd, ind_size);
	hipMalloc((void**)&datad, data_size);
	hipMalloc((void**)&vectord, vecor_size);
	hipMalloc((void**)&resultd, result_size);

	//przeslanie do CUDY

	hipMemcpy(ptrd, ptr, ptr_size, hipMemcpyHostToDevice);
	hipMemcpy(indd, ind, ind_size, hipMemcpyHostToDevice);
	hipMemcpy(datad, data, data_size, hipMemcpyHostToDevice);
	hipMemcpy(vectord, vector, vecor_size, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, result_size, hipMemcpyHostToDevice);

	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(grid_number);

	// mnozenie
	
	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	 
         multiplyCRS <<< dimGrid, dimBlock >>> (ptrd, indd, datad, vectord, resultd, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	printf("Time difference: %f ms \n", time);
               
	// odsylanie
	hipMemcpy(result, resultd, result_size, hipMemcpyDeviceToHost);

	// uwolnienie
	
	hipFree(ptrd);
	hipFree(indd);
	hipFree(datad);
	
	/*
	double* result2 = m.getResult();
	for (int i = 0; i < N; i++){
		if (result2[i] != 0){
			printf(" %f ---- %d ---- %f  \n", result[i]/100, i, result2[i]);
		}
	}

	*/
	printf("ilosc wierszy: %d  \n", N);
	printf("grid_number: %d  \n", grid_number);
	printf("threads_number: %d  \n", thread_number);
}