#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace std;


__global__
void multiply(double *mtx, double *vec, double *res, int N)
{
	for (int i = 0; i < N; i++){
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N){
			res[idx] += mtx[idx * N + i] * vec[i];
		}
	}
}


__global__
void multiplyCompressed(double *mtx, double *vec, double *res, int rows, int columns)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int positionInMatrix = idx;

	if (idx < rows){
		for (int column = columns - 1; column >= 0; column--){
			if (positionInMatrix >= 0){
				if (column < columns - 1){
					res[idx] += vec[positionInMatrix] * mtx[idx*columns + column];
					res[positionInMatrix] += vec[idx] * mtx[idx*columns + column];
				}
				else{
					res[idx] += vec[positionInMatrix] * mtx[idx*columns + column];
				}
				positionInMatrix--;
			}
		}
	}
}

int main()
{
	Matrix m;
	m.readFromMtx("bcsstk27.mtx");
	//m.loadMatrix("matrix.txt");
	m.generateNewVector();
	m.multiplyMatrix();

	int block_size = 256;
	int N = m.getSize();
	int grid_number = ceil(N / (float)block_size);
	int thread_number = grid_number * block_size;
	double *matrixd;
	double *vectord;
	double *resultd;
	double *matrix = m.getMatrix();
	double *vector = m.getVector();
	double *result = new double[N];
	//for (int i = 0; i < N*N; i++){
	//	if (matrix[i] != 0){
	//		printf("%d: %f \n", i, matrix[i]);
	//	}
	//}
	fill(result, result + N, 0.);

	const int martix_size = N*N*sizeof(double);
	const int vecor_size = N*sizeof(double);
	const int result_size = N*sizeof(double);


	hipMalloc((void**)&matrixd, martix_size);
	hipMalloc((void**)&vectord, vecor_size);
	hipMalloc((void**)&resultd, result_size);

	hipMemcpy(matrixd, matrix, martix_size, hipMemcpyHostToDevice);
	hipMemcpy(vectord, vector, vecor_size, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, result_size, hipMemcpyHostToDevice);

	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(grid_number);
	multiply << <dimGrid, dimBlock >> >(matrixd, vectord, resultd, N);
	hipMemcpy(result, resultd, result_size, hipMemcpyDeviceToHost);
	hipFree(matrixd);
	hipFree(vectord);
	hipFree(resultd);
	double* result2 = m.getResult();
	for (int i = 0; i < N; i++){
		if (result2[i] != 0){
			printf(" %f ---- %d ---- %f  \n", result[i], i, result2[i]);
		}
	}
	printf("ilosc wierszy: %d  \n", N);
	printf("grid_number: %d  \n", grid_number);
	printf("threads_number: %d  \n", thread_number);
}

int main()
{
	Matrix m;
	m.readFromMtx("bcsstk28.mtx");
	//m.loadMatrix("matrix.txt");
	m.generateNewVector();
	m.diagonalCompression();
	m.multiplayDiagonal();


	int block_size = 256;
	int rows = m.getSize();
	int columns = m.getCompressedColumnSize();
	printf("rows: %d ", rows);
	printf("columns: %d ", columns);

	int grid_number = ceil(rows / (float)block_size);
	int thread_number = grid_number * block_size;

	double *matrixd;
	double *vectord;
	double *resultd;
	double *matrix = m.getCompressedMatrix();
	double *vector = m.getVector();
	double *result = new double[rows];

	//for (int i = 0; i < rows*columns; i++){
	//	printf(" %f ", matrix[i]);
	//	if ((i+1)%columns == 0){
	//		printf("\n");
	//	}
	//}

	fill(result, result + rows, 0.);
	const int martix_size = rows*columns*sizeof(double);
	const int vecor_size = rows*sizeof(double);
	const int result_size = rows*sizeof(double);


	hipMalloc((void**)&matrixd, martix_size);
	hipMalloc((void**)&vectord, vecor_size);
	hipMalloc((void**)&resultd, result_size);

	hipMemcpy(matrixd, matrix, martix_size, hipMemcpyHostToDevice);
	hipMemcpy(vectord, vector, vecor_size, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, result_size, hipMemcpyHostToDevice);

	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(grid_number);
	multiplyCompressed << <dimGrid, dimBlock >> >(matrixd, vectord, resultd, rows, columns);
	hipMemcpy(result, resultd, result_size, hipMemcpyDeviceToHost);
	hipFree(matrixd);
	hipFree(vectord);
	hipFree(resultd);

	double* result2 = m.getCompressedResult();
	for (int i = 0; i < rows; i++){
		if (result2[i] != 0){
			printf(" %f ---- %d ---- %f  \n", result[i], i, result2[i]);
		}
	}
	printf("ilosc wierszy: %d  \n", rows);
	printf("grid_number: %d  \n", grid_number);
	printf("threads_number: %d  \n", thread_number);
}