#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Matrix.h"
#include <windows.h>
#include <vector>

using namespace std;

#define TIMER_INIT \
    LARGE_INTEGER frequency; \
    LARGE_INTEGER t1,t2; \
    double elapsedTime; \
    QueryPerformanceFrequency(&frequency);

#define TIMER_START QueryPerformanceCounter(&t1);

#define TIMER_STOP \
    QueryPerformanceCounter(&t2); \
    elapsedTime=(float)(t2.QuadPart-t1.QuadPart)/frequency.QuadPart; \


__global__
void multiplyCRS(int *ptr, int *ind, double *data, double *vec, double *res, int N) {
	int row = blockDim.x * blockIdx.x + threadIdx.x;

	if ( row < N ){
		double suma = 0;
		for (int jj = ptr[row]; jj < ptr[row + 1]; jj++){
			int j = ind[jj];
			suma += data[jj] * vec[j];
		}
		res[row] += suma;
	}
}




int main()
{
	Matrix m;
	m.readFromMtx("bcsstk27.mtx");
	m.generateNewVector();
	m.multiplyMatrix();
	m.modifiedSparseCompression();
	TIMER_INIT;

	int block_size = 256;
	int N = m.getSize();
	int I = m.getIndSize();
	int P = m.getPtrSize();
	int D = m.getDataSize();
	int grid_number = ceil(N / (float)block_size);
	int thread_number = grid_number * block_size;

	double summary = 0.0;
	double *vectord;
	double *resultd;
	int *ptrd; 
	int *indd; 
	double *datad;
	vector <double> check;
	double *vector = m.getVector();
	double *result = new double[N];
	int *ptr = m.getPtr();
	int *ind = m.getInd();
	double *data = m.getData();

	fill(result, result + N, 0.);

	// rozmiar
	const int vecor_size = N*sizeof(double);
	const int result_size = N*sizeof(double);
	const int data_size = D*sizeof(double);
	const int ptr_size = P*sizeof(int);
	const int ind_size = I*sizeof(int);

	// zalokowanie
	
	hipMalloc((void**)&ptrd, ptr_size);
	hipMalloc((void**)&indd, ind_size);
	hipMalloc((void**)&datad, data_size);
	hipMalloc((void**)&vectord, vecor_size);
	hipMalloc((void**)&resultd, result_size);

	//przeslanie do CUDY

	hipMemcpy(ptrd, ptr, ptr_size, hipMemcpyHostToDevice);
	hipMemcpy(indd, ind, ind_size, hipMemcpyHostToDevice);
	hipMemcpy(datad, data, data_size, hipMemcpyHostToDevice);
	hipMemcpy(vectord, vector, vecor_size, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, result_size, hipMemcpyHostToDevice);

	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(grid_number);

	// mnozenie
	

	 cout<<"CRS CUDA: ";
            for(int i = 0; i < 100; i++){
                TIMER_START
                   multiplyCRS <<< dimGrid, dimBlock >>> (ptrd, indd, datad, vectord, resultd, N);
                TIMER_STOP
                check.push_back(elapsedTime);
            }

            for(int i = 0; i < check.size(); i++){
                summary+=check[i];
            }

     

            cout<<summary/100<<" sekund"<<endl;
	// odsylanie
	hipMemcpy(result, resultd, result_size, hipMemcpyDeviceToHost);

	// uwolnienie
	
	hipFree(ptrd);
	hipFree(indd);
	hipFree(datad);
	
	/*
	double* result2 = m.getResult();
	for (int i = 0; i < N; i++){
		if (result2[i] != 0){
			printf(" %f ---- %d ---- %f  \n", result[i]/100, i, result2[i]);
		}
	}

	*/
	printf("ilosc wierszy: %d  \n", N);
	printf("grid_number: %d  \n", grid_number);
	printf("threads_number: %d  \n", thread_number);
}