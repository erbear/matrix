#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace std;


__global__
void multiply(double *mtx, double *vec, double *res, int N)
{
	for (int i = 0; i < N; i++){
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N){
			res[idx] += mtx[idx * N + i] * vec[i];
		}
	}
}

int main()
{
	Matrix m;
	m.readFromMtx("bcsstk27.mtx");
	//m.loadMatrix("matrix.txt");
	m.generateNewVector();
	m.multiplyMatrix();

	int block_size = 256;
	int N = m.getSize();
	int grid_number = ceil(N / (float)block_size);
	int thread_number = grid_number * block_size;
	double *matrixd;
	double *vectord;
	double *resultd;
	double *matrix = m.getMatrix();
	double *vector = m.getVector();
	double *result = new double[N];
	//for (int i = 0; i < N*N; i++){
	//	if (matrix[i] != 0){
	//		printf("%d: %f \n", i, matrix[i]);
	//	}
	//}
	fill(result, result + N, 0.);

	const int martix_size = N*N*sizeof(double);
	const int vecor_size = N*sizeof(double);
	const int result_size = N*sizeof(double);


	hipMalloc((void**)&matrixd, martix_size);
	hipMalloc((void**)&vectord, vecor_size);
	hipMalloc((void**)&resultd, result_size);

	hipMemcpy(matrixd, matrix, martix_size, hipMemcpyHostToDevice);
	hipMemcpy(vectord, vector, vecor_size, hipMemcpyHostToDevice);
	hipMemcpy(resultd, result, result_size, hipMemcpyHostToDevice);

	dim3 dimBlock(block_size, 1);
	dim3 dimGrid(grid_number);
	multiply << <dimGrid, dimBlock >> >(matrixd, vectord, resultd, N);
	hipMemcpy(result, resultd, result_size, hipMemcpyDeviceToHost);
	hipFree(matrixd);
	hipFree(vectord);
	hipFree(resultd);
	double* result2 = m.getResult();
	for (int i = 0; i < N; i++){
		if (result2[i] != 0){
			printf(" %f ---- %d ---- %f  \n", result[i], i, result2[i]);
		}
	}
	printf("ilosc wierszy: %d  \n", N);
	printf("grid_number: %d  \n", grid_number);
	printf("threads_number: %d  \n", thread_number);
}